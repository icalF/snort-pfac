#include "icalcu.h"
#include <hip/hip_runtime.h>

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n",  devProp.totalConstMem);
    printf("Texture alignment:             %u\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int main(int argc, char **argv) 
{
    ptr_int h_array, d_array;
    
    int n = NUM_VALS;
    int err;

    // hipDeviceProp_t props;
    // hipGetDeviceProperties(&props, 0);
    // printDevProp(props);

    // scanf("%d",&n);
    
    h_array = (ptr_int) malloc(n * sizeof(int));

    for(int i = 0; i < n; i++) 
    {
        h_array[i] = rand() % 452986;
    }

    HANDLE_ERROR( hipMalloc(&d_array, n * sizeof(int)) );
    HANDLE_ERROR( hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice) );

    sort(d_array, n);

    // reduce<<<BLOCKS, THREADS, SHM_SIZE>>>(d_array);
    // HANDLE_ERROR( hipDeviceSynchronize() );

    // reduce<<<1, THREADS, SHM_SIZE>>>(d_array);
    
    // HANDLE_ERROR( hipDeviceSynchronize() );

    HANDLE_ERROR( hipMemcpy(h_array, d_array, n * sizeof(int), hipMemcpyDeviceToHost) );

    for(int i = 0; i < n; i++) 
    {
        printf("%d ", h_array[i]);
    }
    // puts(h_array[0] ? "TRUE" : "False");
    
    hipFree(d_array);
    free(h_array);    
}

void sort (ptr_int data, int len)
{   
    hipEvent_t start, stop;
    float time;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    localSort<<<BLOCKS, THREADS, SHM_SIZE>>>(data);
    HANDLE_ERROR( hipDeviceSynchronize() );

    for (int block = THREADS << 1; block <= len; block <<= 1)
    {
        /* Global merging */
        // for (int innerBlock = block; innerBlock > THREADS; innerBlock >>= 1)
        // {
        //     globalMerge<<<BLOCKS, THREADS>>>(data, innerBlock, block);          
        //     HANDLE_ERROR( hipDeviceSynchronize() );
        // }

        // /* Local merging */        
        // localMerge<<<BLOCKS, THREADS, SHM_SIZE>>>(data, block);
        // HANDLE_ERROR( hipDeviceSynchronize() );
    }
    
    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    
    printf("%lf ms\n", time);
}