#include "icalcu.h"
#include <hip/hip_runtime.h>

int main(int argc, char **argv) 
{
    ptr_int h_array, d_array;
    int n = NUM_VALS;
    int err;

    // scanf("%d",&n);
    
    h_array = (ptr_int) malloc(n * sizeof(int));
    HANDLE_ERROR( hipMalloc(&d_array, n * sizeof(int)) );

    // for(int i = 0; i < n; i++) 
    // {
    //     scanf("%d ", &h_array[i]);        // rand() % 2;
    // }

    // HANDLE_ERROR( hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice) );

    // HANDLE_ERROR( hipEventCreate(&start) );
    // HANDLE_ERROR( hipEventCreate(&stop) );
    // HANDLE_ERROR( hipEventRecord(start, 0) );

    // reduce<<<BLOCKS, THREADS, SHM_SIZE>>>(d_array);
    // HANDLE_ERROR( hipDeviceSynchronize() );

    // HANDLE_ERROR( hipEventRecord(stop, 0) );
    // HANDLE_ERROR( hipEventSynchronize(stop) );
    // HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );

    // reduce<<<1, THREADS, SHM_SIZE>>>(d_array);
    sort<<<BLOCKS, THREADS, SHM_SIZE>>>(d_array);
    // HANDLE_ERROR( hipDeviceSynchronize() );

    // HANDLE_ERROR( hipMemcpy(h_array, d_array, n * sizeof(int), hipMemcpyDeviceToHost) );

    // for(int i = 0; i < n; i++) 
    // {
    //     printf("%d ", h_array[i]);
    // }
    // puts(h_array[0] ? "TRUE" : "False");

    hipFree(d_array);
    free(h_array);    
}