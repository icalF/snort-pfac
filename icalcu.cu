#include "icalcu.h"
#include <hip/hip_runtime.h>

int main(int argc, char **argv) 
{
    ptr_int h_array, d_array;
    
    int n = NUM_VALS;
    int err;

    // scanf("%d",&n);
    
    h_array = (ptr_int) malloc(n * sizeof(int));

    for(int i = 0; i < n; i++) 
    {
        h_array[i] = rand() % 452986;
    }

    HANDLE_ERROR( hipMalloc(&d_array, n * sizeof(int)) );
    HANDLE_ERROR( hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice) );

    sort(d_array, n);

    // reduce<<<BLOCKS, THREADS, SHM_SIZE>>>(d_array);
    // HANDLE_ERROR( hipDeviceSynchronize() );

    // reduce<<<1, THREADS, SHM_SIZE>>>(d_array);
    
    // HANDLE_ERROR( hipDeviceSynchronize() );

    HANDLE_ERROR( hipMemcpy(h_array, d_array, n * sizeof(int), hipMemcpyDeviceToHost) );

    // for(int i = 0; i < n; i++) 
    // {
    //     printf("%d ", h_array[i]);
    // }
    // puts(h_array[0] ? "TRUE" : "False");
    
    hipFree(d_array);
    free(h_array);    
}

void sort (ptr_int data, int len)
{   
    hipEvent_t start, stop;
    float time;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    localSort<<<BLOCKS, THREADS, SHM_SIZE>>>(data);
    HANDLE_ERROR( hipDeviceSynchronize() );

    for (int block = THREADS << 1; block <= len; block <<= 1)
    {
        /* Aligning */
        globalAlign<<<BLOCKS, THREADS>>>(data, block);
        HANDLE_ERROR( hipDeviceSynchronize() );

        /* Global merging */
        for (int innerBlock = block >> 1; innerBlock > THREADS; innerBlock >>= 1)
        {
            globalMerge<<<BLOCKS, THREADS>>>(data, innerBlock);          
            HANDLE_ERROR( hipDeviceSynchronize() );
        }

        /* Local merging */
        for (int innerBlock = THREADS; innerBlock > 1; innerBlock >>= 1)
        {
            localMerge<<<BLOCKS, THREADS, SHM_SIZE>>>(data, innerBlock);
            HANDLE_ERROR( hipDeviceSynchronize() );
        }
    }
    
    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    
    printf("%lf ms\n", time);
}