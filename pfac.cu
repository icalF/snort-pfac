#include "cuda_utils.h"

#include "pfac.h"
#include "pfac_match.h"
#include "pfac_table.h"

using namespace std;

static inline void ConvertCaseEx (unsigned char *d, unsigned char *s, int m)
{
    int i;
    for (i = 0; i < m; i++)
    {
        d[i] = xlatcase[s[i]];
    }
}

 PFAC_status_t  PFAC_destroy( PFAC_handle_t handle )
{
    if ( NULL == handle ){
        return PFAC_STATUS_INVALID_HANDLE ;
    }

    PFAC_freeResource( handle ) ;

    free( handle ) ;

    return PFAC_STATUS_SUCCESS ;
}

void  PFAC_freeResource( PFAC_handle_t handle )
{
    // resource of patterns
    if ( NULL != handle->rowPtr ){
        free( handle->rowPtr );
        handle->rowPtr = NULL ;
    }
    
    if ( NULL != handle->valPtr ){
        free( handle->valPtr );
        handle->valPtr = NULL ;
    }

    if ( NULL != handle->patternLen_table ){
        free( handle->patternLen_table ) ;
        handle->patternLen_table = NULL ;
    }
    
    if ( NULL != handle->patternID_table ){
        free( handle->patternID_table );
        handle->patternID_table = NULL ;
    }
    
    if ( NULL != handle->table_compact ){
        delete  handle->table_compact ;
        handle->table_compact = NULL ;
    }

    PFAC_freeTable( handle );
 
    handle->isPatternsReady = false ;
}

void  PFAC_freeTable( PFAC_handle_t handle )
{
    if ( NULL != handle->h_PFAC_table ){
        free( handle->h_PFAC_table ) ;
        handle->h_PFAC_table = NULL ;
    }

    if ( NULL != handle->h_hashRowPtr ){
        free( handle->h_hashRowPtr );
        handle->h_hashRowPtr = NULL ;   
    }
    
    if ( NULL != handle->h_hashValPtr ){
        free( handle->h_hashValPtr );
        handle->h_hashValPtr = NULL ;   
    }
    
    if ( NULL != handle->h_tableOfInitialState){
        free(handle->h_tableOfInitialState);
        handle->h_tableOfInitialState = NULL ; 
    }
    
    // free device resource
    if ( NULL != handle->d_PFAC_table ){
        hipFree(handle->d_PFAC_table);
        handle->d_PFAC_table= NULL ;
    }
    
    if ( NULL != handle->d_hashRowPtr ){
        hipFree( handle->d_hashRowPtr );
        handle->d_hashRowPtr = NULL ;
    }

    if ( NULL != handle->d_hashValPtr ){
        hipFree( handle->d_hashValPtr );
        handle->d_hashValPtr = NULL ;   
    }
    
    if ( NULL != handle->d_tableOfInitialState ){
        hipFree(handle->d_tableOfInitialState);
        handle->d_tableOfInitialState = NULL ;
    }   
}

/* warpper for pthread_mutex_lock and pthread_mutex_unlock */
mutex  __pfac_tex_mutex;    

PFAC_status_t PFAC_tex_mutex_lock(void)
{
    try
    {
        __pfac_tex_mutex.lock();
    }
    catch (const system_error &e)
    {
        return PFAC_STATUS_MUTEX_ERROR;
    }

    return PFAC_STATUS_SUCCESS;
}

PFAC_status_t PFAC_tex_mutex_unlock(void)
{
    try
    {
        __pfac_tex_mutex.unlock();
    }
    catch (const system_error &e)
    {
        return PFAC_STATUS_MUTEX_ERROR;
    }

    return PFAC_STATUS_SUCCESS;
}

PFAC_status_t  PFAC_create( PFAC_handle_t handle )
{
    handle = (PFAC_handle_t) malloc( sizeof(PFAC_STRUCT) ) ;

    if ( NULL == handle ){
        return PFAC_STATUS_ALLOC_FAILED ;
    }

    memset( handle, 0, sizeof(PFAC_STRUCT) ) ;

    int device ;
    hipError_t cuda_status = hipGetDevice( &device ) ;
    if ( hipSuccess != cuda_status ){
        return (PFAC_status_t)cuda_status ;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    PFAC_PRINTF("major = %d, minor = %d, name=%s\n", deviceProp.major, deviceProp.minor, deviceProp.name );

    int device_no = 10*deviceProp.major + deviceProp.minor ;
    
    handle->device_no = device_no ;

    // Find entry point of PFAC_kernel
    handle->kernel_ptr = (PFAC_kernel_protoType) PFAC_kernel_timeDriven_wrapper;
    if ( NULL == handle->kernel_ptr ){
        PFAC_PRINTF("Error: cannot load PFAC_kernel_timeDriven_wrapper, error = %s\n", "" );
        return PFAC_STATUS_INTERNAL_ERROR ;
    }

    return PFAC_STATUS_SUCCESS ;
}

PFAC_STRUCT * pfacNew (void (*userfree)(void *p),
        void (*optiontreefree)(void **p),
        void (*neg_list_free)(void **p))
{
    PFAC_handle_t handle;
    PFAC_status_t status = PFAC_create( handle );

    if ( status != PFAC_STATUS_SUCCESS )
    {
        PFAC_PRINTF("Error: cannot initialize handler, error = %s\n", PFAC_getErrorString(status));
        return NULL;
    }

    init_xlatcase();

    handle->userfree              = userfree;
    handle->optiontreefree        = optiontreefree;
    handle->neg_list_free         = neg_list_free;

    return (PFAC_STRUCT *) handle;
}

void pfacFree ( PFAC_STRUCT * pfac )
{
    PFAC_handle_t handle = (PFAC_handle_t) pfac;
    PFAC_status_t status = PFAC_destroy( handle ) ;
    if ( status != PFAC_STATUS_SUCCESS )
    {
        PFAC_PRINTF("Error: cannot deinitialize handler, error = %s\n", PFAC_getErrorString(status));
    }
}

int pfacAddPattern ( PFAC_STRUCT * p, unsigned char *pat, int n, int nocase,
                     int offset, int depth, int negative, void * id, int iid )
{
    PFAC_PATTERN * plist;
    plist = (PFAC_PATTERN *) calloc (1, sizeof (PFAC_PATTERN));
    plist->patrn = (unsigned char *) calloc (1, n);
    ConvertCaseEx (plist->patrn, pat, n);
    plist->casepatrn = (unsigned char *) calloc (1, n);
    memcpy (plist->casepatrn, pat, n);

    plist->udata = (PFAC_USERDATA *) calloc (1, sizeof (PFAC_USERDATA));
    plist->udata->ref_count = 1;
    plist->udata->id = id;

    plist->n = n;
    plist->nocase = nocase;
    plist->negative = negative;
    plist->offset = offset;
    plist->depth = depth;
    plist->iid = iid;
    plist->next = p->pfacPatterns;
    p->pfacPatterns = plist;
    p->numOfPatterns++;
    p->max_numOfStates += n + 1;
    return 0;
}


int pfacCompile ( PFAC_STRUCT * pfac,
        int (*build_tree)(void * id, void **existing_tree),
        int (*neg_list_func)(void *id, void **list))
{
    int max_numOfStates = pfac->max_numOfStates;

    // Allocate a buffer to contains all patterns
    pfac->valPtr = (char*)malloc(sizeof(char)*max_numOfStates);
    if (NULL == pfac->valPtr) {
        return PFAC_STATUS_ALLOC_FAILED;
    }

    /* Copy all patterns into the buffer */
    PFAC_PATTERN *plist;
    char *offset;
    for (plist = pfac->pfacPatterns, offset = pfac->valPtr + 1;
         plist != NULL; 
         offset += plist->n + 1, plist = plist->next)
    {
        memcpy(offset, plist->patrn, plist->n);
    }

    char *buffer = pfac->valPtr;
    vector< struct patternEle > rowIdxArray;
    vector<int>  patternLenArray;
    int len;

    struct patternEle pEle;

    pEle.patternString = buffer;
    pEle.patternID = 1;

    rowIdxArray.push_back(pEle);
    len = 0;
    for (int i = 0; i < max_numOfStates; i++) {
        if (( '\n' == buffer[i] ) || ( '\0' == buffer[i]) ) {
            if (( i > 0 ) && ( '\n' != buffer[i - 1] ) && ( '\0' != buffer[i - 1] )) { // non-empty line
                patternLenArray.push_back(len);
                pEle.patternString = buffer + i + 1; // start of next pattern
                pEle.patternID = rowIdxArray.size() + 1; // ID of next pattern
                rowIdxArray.push_back(pEle);
            }
            len = 0;
        }
        else {
            len++;
        }
    }

    // rowIdxArray.size()-1 = number of patterns
    // sort patterns by lexicographic order
    sort(rowIdxArray.begin(), rowIdxArray.begin() + pfac->numOfPatterns, pattern_cmp_functor());

    pfac->rowPtr = (char**)malloc(sizeof(char*)*rowIdxArray.size());
    pfac->patternID_table = (int*)malloc(sizeof(int)*rowIdxArray.size());
    // suppose there are k patterns, then size of patternLen_table is k+1
    // because patternLen_table[0] is useless, valid data starts from
    // patternLen_table[1], up to patternLen_table[k]
    pfac->patternLen_table = (int*)malloc(sizeof(int)*rowIdxArray.size());
    if ((NULL == pfac->rowPtr) ||
        (NULL == pfac->patternID_table) ||
        (NULL == pfac->patternLen_table))
    {
        return PFAC_STATUS_ALLOC_FAILED;
    }

    // Compute f(final state) = patternID
    for (int i = 0; i < (rowIdxArray.size() - 1); i++) {
        pfac->rowPtr[i] = rowIdxArray[i].patternString;
        pfac->patternID_table[i] = rowIdxArray[i].patternID; // pattern number starts from 1
    }

    // although patternLen_table[0] is useless, in order to avoid errors from valgrind
    // we need to initialize patternLen_table[0]
    pfac->patternLen_table[0] = 0;
    for (int i = 0; i < (rowIdxArray.size() - 1); i++) {
        // pattern (*rowPtr)[i] is terminated by character '\n'
        // pattern ID starts from 1, so patternID = i+1
        pfac->patternLen_table[i + 1] = patternLenArray[i];
    }

    return PFAC_STATUS_SUCCESS;
}

int pfacSearch ( PFAC_STRUCT * pfac,unsigned char * T, int n, 
        int (*Match)(void * id, void *tree, int index, void *data, void *neg_list),
        void * data, int* current_state )
{
    int *h_matched_result = (int *) malloc ( n * sizeof(int) );
    int nfound = 0;
    PFAC_handle_t handle = (PFAC_handle_t) pfac;

    PFAC_status_t status = PFAC_matchFromHost( handle, (char *) T, n, h_matched_result ) ;

    if ( status != PFAC_STATUS_SUCCESS ) {
        PFAC_PRINTF("Error: fails to PFAC_matchFromHost, %s\n", PFAC_getErrorString(status) );
        return 0;
    }

    for (int i = 0; i < n; ++i)
    {
        nfound += (h_matched_result[i] > 0);
    }
    return nfound;
}

int pfacPrintDetailInfo(PFAC_STRUCT * p)
{
    if(p)
        p = p;
    return 0;
}

int pfacPrintSummaryInfo(void)
{
    // SPFAC_STRUCT2 * p = &summary.spfac;

    // if( !summary.num_states )
    //     return;

    // PFAC_PRINTF("+--[Pattern Matcher:Aho-Corasick Summary]----------------------\n");
    // PFAC_PRINTF("| Alphabet Size    : %d Chars\n",p->spfacAlphabetSize);
    // PFAC_PRINTF("| Sizeof State     : %d bytes\n",sizeof(acstate_t));
    // PFAC_PRINTF("| Storage Format   : %s \n",sf[ p->spfacFormat ]);
    // PFAC_PRINTF("| Num States       : %d\n",summary.num_states);
    // PFAC_PRINTF("| Num Transitions  : %d\n",summary.num_transitions);
    // PFAC_PRINTF("| State Density    : %.1f%%\n",100.0*(double)summary.num_transitions/(summary.num_states*p->spfacAlphabetSize));
    // PFAC_PRINTF("| Finite Automatum : %s\n", fsa[p->spfacFSA]);
    // if( max_memory < 1024*1024 )
    //     PFAC_PRINTF("| Memory           : %.2fKbytes\n", (float)max_memory/1024 );
    // else
    //     PFAC_PRINTF("| Memory           : %.2fMbytes\n", (float)max_memory/(1024*1024) );
    // PFAC_PRINTF("+-------------------------------------------------------------\n");

    return 0;
}
